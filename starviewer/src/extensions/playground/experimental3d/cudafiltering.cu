#include "hip/hip_runtime.h"
// tot el que tingui prefix d és del dispositiu

#include "cudafiltering.h"

#include <iostream>

#include <hip/hip_runtime.h>
#include <cutil.h>

#include <vtkImageData.h>


//Round a / b to nearest higher integer value
__device__ __host__ int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}


texture<float, 3> gVolumeTexture;   // el 3r paràmetre pot ser hipReadModeElementType (valor directe) (predeterminat) o hipReadModeNormalizedFloat (valor escalat entre 0 i 1)


__global__ void convolutionXKernel(float *result, float *kernel, int radius, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;

    float sum = 0.0f;

    for (int k = -radius; k <= radius; k++) sum += tex3D(gVolumeTexture, fx + k, fy, fz) * kernel[radius - k];

    uint i = x + y * dims.width + z * dims.width * dims.height;

    result[i] = sum;
}


__global__ void convolutionYKernel(float *result, float *kernel, int radius, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;

    float sum = 0.0f;

    for (int k = -radius; k <= radius; k++) sum += tex3D(gVolumeTexture, fx, fy + k, fz) * kernel[radius - k];

    uint i = x + y * dims.width + z * dims.width * dims.height;

    result[i] = sum;
}


__global__ void convolutionZKernel(float *result, float *kernel, int radius, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;

    float sum = 0.0f;

    for (int k = -radius; k <= radius; k++) sum += tex3D(gVolumeTexture, fx, fy, fz + k) * kernel[radius - k];

    uint i = x + y * dims.width + z * dims.width * dims.height;

    result[i] = sum;
}


__global__ void substractionKernel(float *result, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;
    float value = tex3D(gVolumeTexture, fx, fy, fz);
    uint i = x + y * dims.width + z * dims.width * dims.height;
    result[i] -= value;
}


QVector<float> cfGaussianDifference(vtkImageData *image, float radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int RADIUS = static_cast<int>(ceil(radius));
    const int KERNEL_WIDTH = 2 * RADIUS + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    float kernelSum = 0.0f;
    for (int i = 0; i < KERNEL_WIDTH; i++)
    {
        float distance = (i - radius) / radius;
        kernel[i] = expf(-distance * distance / 2.0f);
        kernelSum += kernel.at(i);
    }
    for (int i = 0; i < KERNEL_WIDTH; i++) kernel[i] /= kernelSum;
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Executar per X
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    convolutionXKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, RADIUS, volumeDataDims);
    CUT_CHECK_ERROR( "convolutionXKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Executar per Y
    convolutionYKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, RADIUS, volumeDataDims);
    CUT_CHECK_ERROR( "convolutionYKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat a l'array
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Executar per Z
    convolutionZKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, RADIUS, volumeDataDims);
    CUT_CHECK_ERROR( "convolutionZKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el volum original un altre cop a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Resta
    substractionKernel<<<blockGrid, threadBlock>>>(dfResult, volumeDataDims);
    CUT_CHECK_ERROR( "substractionKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "gaussian difference: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}
