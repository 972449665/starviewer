#include "hip/hip_runtime.h"
// tot el que tingui prefix d és del dispositiu

#include "cudafiltering.h"

#include <iostream>

#include <hip/hip_runtime.h>
#include <cutil.h>

#include <vtkImageData.h>


//Round a / b to nearest higher integer value
__device__ __host__ int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}


texture<float, 3> gVolumeTexture;   // el 3r paràmetre pot ser hipReadModeElementType (valor directe) (predeterminat) o hipReadModeNormalizedFloat (valor escalat entre 0 i 1)
texture<float, 3> gVolume2Texture;  // el 3r paràmetre pot ser hipReadModeElementType (valor directe) (predeterminat) o hipReadModeNormalizedFloat (valor escalat entre 0 i 1)


__global__ void convolutionXKernel(float *result, float *kernel, int radius, hipExtent dims, bool texture2)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;

    float sum = 0.0f;

    if (!texture2)
    {
        for (int k = -radius; k <= radius; k++) sum += tex3D(gVolumeTexture, fx + k, fy, fz) * kernel[radius - k];
    }
    else
    {
        for (int k = -radius; k <= radius; k++) sum += tex3D(gVolume2Texture, fx + k, fy, fz) * kernel[radius - k];
    }

    uint i = x + y * dims.width + z * dims.width * dims.height;

    result[i] = sum;
}


__global__ void convolutionYKernel(float *result, float *kernel, int radius, hipExtent dims, bool texture2)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;

    float sum = 0.0f;

    if (!texture2)
    {
        for (int k = -radius; k <= radius; k++) sum += tex3D(gVolumeTexture, fx, fy + k, fz) * kernel[radius - k];
    }
    else
    {
        for (int k = -radius; k <= radius; k++) sum += tex3D(gVolume2Texture, fx, fy + k, fz) * kernel[radius - k];
    }

    uint i = x + y * dims.width + z * dims.width * dims.height;

    result[i] = sum;
}


__global__ void convolutionZKernel(float *result, float *kernel, int radius, hipExtent dims, bool texture2)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;

    float sum = 0.0f;

    if (!texture2)
    {
        for (int k = -radius; k <= radius; k++) sum += tex3D(gVolumeTexture, fx, fy, fz + k) * kernel[radius - k];
    }
    else
    {
        for (int k = -radius; k <= radius; k++) sum += tex3D(gVolume2Texture, fx, fy, fz + k) * kernel[radius - k];
    }

    uint i = x + y * dims.width + z * dims.width * dims.height;

    result[i] = sum;
}


__global__ void substractionKernel(float *result, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;
    float value = tex3D(gVolumeTexture, fx, fy, fz);
    uint i = x + y * dims.width + z * dims.width * dims.height;
    result[i] -= value;
}


QVector<float> cfGaussianDifference(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipEvent_t t0, t1;
    float t01 = 0.0f;
    hipEventCreate(&t0);
    hipEventCreate(&t1);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int KERNEL_WIDTH = 2 * radius + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    float kernelSum = 0.0f;
    float sigma = radius / 3.0f;
    for (int i = 0; i < KERNEL_WIDTH; i++)
    {
        float f = static_cast<float>(i - radius) / sigma;
        kernel[i] = expf(-f * f / 2.0f);
        kernelSum += kernel.at(i);
    }
    for (int i = 0; i < KERNEL_WIDTH; i++) kernel[i] /= kernelSum;
    std::cout << "kernel:";
    for (int i = 0; i < KERNEL_WIDTH; i++) std::cout << " " << kernel[i];
    std::cout << std::endl;
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Executar per X
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(t0, 0);
    convolutionXKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    CUT_CHECK_ERROR( "convolutionXKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(t1, 0);
    hipEventSynchronize(t1);
    hipEventElapsedTime(&t01, t0, t1);
    std::cout << "X filter: " << t01 << " ms" << std::endl;

    // Copiar el resultat a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Executar per Y
    hipEventRecord(t0, 0);
    convolutionYKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    CUT_CHECK_ERROR( "convolutionYKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(t1, 0);
    hipEventSynchronize(t1);
    hipEventElapsedTime(&t01, t0, t1);
    std::cout << "Y filter: " << t01 << " ms" << std::endl;

    // Copiar el resultat a l'array
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Executar per Z
    hipEventRecord(t0, 0);
    convolutionZKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    CUT_CHECK_ERROR( "convolutionZKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(t1, 0);
    hipEventSynchronize(t1);
    hipEventElapsedTime(&t01, t0, t1);
    std::cout << "Z filter: " << t01 << " ms" << std::endl;

    // Copiar el volum original un altre cop a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Resta
    hipEventRecord(t0, 0);
    substractionKernel<<<blockGrid, threadBlock>>>(dfResult, volumeDataDims);
    CUT_CHECK_ERROR( "substractionKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(t1, 0);
    hipEventSynchronize(t1);
    hipEventElapsedTime(&t01, t0, t1);
    std::cout << "substraction: " << t01 << " ms" << std::endl;

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "gaussian difference: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(t0);
    hipEventDestroy(t1);

    return result;
}


QVector<float> cfBoxMeanDifference(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int KERNEL_WIDTH = 2 * radius + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    kernel.fill(1.0f / KERNEL_WIDTH);
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Executar per X
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    convolutionXKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    CUT_CHECK_ERROR( "convolutionXKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Executar per Y
    convolutionYKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    CUT_CHECK_ERROR( "convolutionYKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat a l'array
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Executar per Z
    convolutionZKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    CUT_CHECK_ERROR( "convolutionZKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el volum original un altre cop a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Resta
    substractionKernel<<<blockGrid, threadBlock>>>(dfResult, volumeDataDims);
    CUT_CHECK_ERROR( "substractionKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "box mean difference: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}


__global__ void squareKernel(float *result, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;
    float value = tex3D(gVolumeTexture, fx, fy, fz);
    uint i = x + y * dims.width + z * dims.width * dims.height;
    result[i] = value * value;
}


__global__ void finalChebychevKernel(float *result, float *result2, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;
    float value = tex3D(gVolumeTexture, fx, fy, fz);

    uint i = x + y * dims.width + z * dims.width * dims.height;

    float mean = result[i];         // E[Z]

    if (value > mean)
    {
        float squaresMean = result2[i]; // E[Z²]

        float variance = squaresMean - mean * mean;
        float a = value - mean;  // z - E[Z]
        result[i] = variance / (variance + a * a);  // Chebychev inequality
    }
    else result[i] = 1.0f;
}


QVector<float> cfProbabilisticAmbientOcclusionGaussianChebychev(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Calcular volum al quadrat
    squareKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult, volumeDataDims); // generem el volum al quadrat a l'stream 2

    // Mentrestant, al host...

    // Crear un segon array pel volum al quadrat, amb la seva textura corresponent
    hipArray *dVolume2Array;
    hipChannelFormatDesc channelDescVolume2Array = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolume2Array, &channelDescVolume2Array, volumeDataDims) );
    //gVolume2Texture.normalized = false;                     // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolume2Texture.filterMode = hipFilterModePoint;       // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolume2Texture.addressMode[0] = hipAddressModeClamp;  // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolume2Texture.addressMode[1] = hipAddressModeClamp;
    //gVolume2Texture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolume2Texture, dVolume2Array, channelDescVolume2Array) );

    // En aquest temps suposem que ja ha acabat el kernel d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // Copiem el resultat a l'array del volum al quadrat
    hipMemcpy3DParms copyParams2 = {0};
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);   // data, pitch, width, height
    copyParams2.dstArray = dVolume2Array;
    copyParams2.extent = volumeDataDims;
    copyParams2.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) ); // còpia a l'stream 2

    // Reservar espai per l'altre resultat
    float *dfResult2;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult2), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int KERNEL_WIDTH = 2 * radius + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    float kernelSum = 0.0f;
    float sigma = radius / 3.0f;
    for (int i = 0; i < KERNEL_WIDTH; i++)
    {
        float f = static_cast<float>(i - radius) / sigma;
        kernel[i] = expf(-f * f / 2.0f);
        kernelSum += kernel.at(i);
    }
    for (int i = 0; i < KERNEL_WIDTH; i++) kernel[i] /= kernelSum;
    std::cout << "kernel:";
    for (int i = 0; i < KERNEL_WIDTH; i++) std::cout << " " << kernel[i];
    std::cout << std::endl;
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    // Aquest és molt petit, o sigui que el podem fer síncron
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Aquí ja hauria d'haver acabat la còpia d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // A partir d'aquí comença la diversió:
    // Executarem els dos filtratges en paral·lel, en streams diferents, perquè mentre un filtra l'altre copiï memòria i viceversa.
    // Així podem aconseguir una execució més ràpida (en teoria).
    // Som-hi doncs...

    // Executar per X1
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per X2
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult2), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);  // data, pitch, width, height
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Y1
    hipStreamSynchronize(stream1);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Y2
    hipStreamSynchronize(stream2);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Z1
    hipStreamSynchronize(stream1);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Z2
    hipStreamSynchronize(stream2);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el volum original un altre cop a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyHostToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // aquesta la fem síncrona
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Ara ja podem fer la passada final
    finalChebychevKernel<<<blockGrid, threadBlock>>>(dfResult, dfResult2, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipFree(dfResult2) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolume2Texture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );
    CUDA_SAFE_CALL( hipFreeArray(dVolume2Array) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "box mean chebychev: " << elapsedTime << " ms" << std::endl;

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}


QVector<float> cfProbabilisticAmbientOcclusionBoxMeanChebychev(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Calcular volum al quadrat
    squareKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult, volumeDataDims); // generem el volum al quadrat a l'stream 2

    // Mentrestant, al host...

    // Crear un segon array pel volum al quadrat, amb la seva textura corresponent
    hipArray *dVolume2Array;
    hipChannelFormatDesc channelDescVolume2Array = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolume2Array, &channelDescVolume2Array, volumeDataDims) );
    //gVolume2Texture.normalized = false;                     // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolume2Texture.filterMode = hipFilterModePoint;       // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolume2Texture.addressMode[0] = hipAddressModeClamp;  // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolume2Texture.addressMode[1] = hipAddressModeClamp;
    //gVolume2Texture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolume2Texture, dVolume2Array, channelDescVolume2Array) );

    // En aquest temps suposem que ja ha acabat el kernel d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // Copiem el resultat a l'array del volum al quadrat
    hipMemcpy3DParms copyParams2 = {0};
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);   // data, pitch, width, height
    copyParams2.dstArray = dVolume2Array;
    copyParams2.extent = volumeDataDims;
    copyParams2.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) ); // còpia a l'stream 2

    // Reservar espai per l'altre resultat
    float *dfResult2;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult2), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int KERNEL_WIDTH = 2 * radius + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    kernel.fill(1.0f / KERNEL_WIDTH);
    std::cout << "kernel:";
    for (int i = 0; i < KERNEL_WIDTH; i++) std::cout << " " << kernel[i];
    std::cout << std::endl;
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    // Aquest és molt petit, o sigui que el podem fer síncron
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Aquí ja hauria d'haver acabat la còpia d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // A partir d'aquí comença la diversió:
    // Executarem els dos filtratges en paral·lel, en streams diferents, perquè mentre un filtra l'altre copiï memòria i viceversa.
    // Així podem aconseguir una execució més ràpida (en teoria).
    // Som-hi doncs...

    // Executar per X1
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per X2
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult2), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);  // data, pitch, width, height
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Y1
    hipStreamSynchronize(stream1);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Y2
    hipStreamSynchronize(stream2);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Z1
    hipStreamSynchronize(stream1);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Z2
    hipStreamSynchronize(stream2);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el volum original un altre cop a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyHostToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // aquesta la fem síncrona
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Ara ja podem fer la passada final
    finalChebychevKernel<<<blockGrid, threadBlock>>>(dfResult, dfResult2, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipFree(dfResult2) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolume2Texture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );
    CUDA_SAFE_CALL( hipFreeArray(dVolume2Array) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "box mean chebychev: " << elapsedTime << " ms" << std::endl;

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}


__global__ void finalGaussianKernel(float *result, float *result2, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;
    float value = tex3D(gVolumeTexture, fx, fy, fz);

    uint i = x + y * dims.width + z * dims.width * dims.height;

    float mean = result[i];         // E[Z]
    float squaresMean = result2[i]; // E[Z²]
    float variance = squaresMean - mean * mean;
    // Fórmules tretes de:
    // http://en.wikipedia.org/wiki/Gaussian_distribution#Cumulative_distribution_function
    // http://en.wikipedia.org/wiki/Q-function
    // P(Z >= z) = 1/2 * erfc((z - mean) / sqrt(2 * variance))
    result[i] = 0.5f * erfcf((value - mean) / sqrtf(2.0f * variance));
}


QVector<float> cfProbabilisticAmbientOcclusionGaussian(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Calcular volum al quadrat
    squareKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult, volumeDataDims); // generem el volum al quadrat a l'stream 2

    // Mentrestant, al host...

    // Crear un segon array pel volum al quadrat, amb la seva textura corresponent
    hipArray *dVolume2Array;
    hipChannelFormatDesc channelDescVolume2Array = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolume2Array, &channelDescVolume2Array, volumeDataDims) );
    //gVolume2Texture.normalized = false;                     // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolume2Texture.filterMode = hipFilterModePoint;       // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolume2Texture.addressMode[0] = hipAddressModeClamp;  // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolume2Texture.addressMode[1] = hipAddressModeClamp;
    //gVolume2Texture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolume2Texture, dVolume2Array, channelDescVolume2Array) );

    // En aquest temps suposem que ja ha acabat el kernel d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // Copiem el resultat a l'array del volum al quadrat
    hipMemcpy3DParms copyParams2 = {0};
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);   // data, pitch, width, height
    copyParams2.dstArray = dVolume2Array;
    copyParams2.extent = volumeDataDims;
    copyParams2.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) ); // còpia a l'stream 2

    // Reservar espai per l'altre resultat
    float *dfResult2;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult2), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int KERNEL_WIDTH = 2 * radius + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    float kernelSum = 0.0f;
    float sigma = radius / 3.0f;
    for (int i = 0; i < KERNEL_WIDTH; i++)
    {
        float f = static_cast<float>(i - radius) / sigma;
        kernel[i] = expf(-f * f / 2.0f);
        kernelSum += kernel.at(i);
    }
    for (int i = 0; i < KERNEL_WIDTH; i++) kernel[i] /= kernelSum;
    std::cout << "kernel:";
    for (int i = 0; i < KERNEL_WIDTH; i++) std::cout << " " << kernel[i];
    std::cout << std::endl;
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    // Aquest és molt petit, o sigui que el podem fer síncron
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Aquí ja hauria d'haver acabat la còpia d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // A partir d'aquí comença la diversió:
    // Executarem els dos filtratges en paral·lel, en streams diferents, perquè mentre un filtra l'altre copiï memòria i viceversa.
    // Així podem aconseguir una execució més ràpida (en teoria).
    // Som-hi doncs...

    // Executar per X1
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per X2
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult2), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);  // data, pitch, width, height
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Y1
    hipStreamSynchronize(stream1);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Y2
    hipStreamSynchronize(stream2);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Z1
    hipStreamSynchronize(stream1);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Z2
    hipStreamSynchronize(stream2);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el volum original un altre cop a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyHostToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // aquesta la fem síncrona
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Ara ja podem fer la passada final
    finalGaussianKernel<<<blockGrid, threadBlock>>>(dfResult, dfResult2, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipFree(dfResult2) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolume2Texture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );
    CUDA_SAFE_CALL( hipFreeArray(dVolume2Array) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "box mean chebychev: " << elapsedTime << " ms" << std::endl;

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}
